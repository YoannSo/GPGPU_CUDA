#include "hip/hip_runtime.h"

#include "WorkOnGPU.hpp"

int* workOnGpu(vector<int> rgb){
    float totalTime=0;
    int HSVsize=rgb.size()/3;
    vector<float> vectorh=vector<float>();
    vector<float> vectors=vector<float>();
    vector<float> vectorv=vector<float>();
    int nbPixels=rgb.size()/3+1;
    int nbBlocks;
    float* h=new float[HSVsize];
    float* s=new float[HSVsize];
    float* v=new float[HSVsize];
    vector<int> newRGBVector=vector<int>();
    int* dev_histo=new int[256];
    int* dev_repartition=new int[256];
    ChronoGPU chrGPU;

    cout << "============================================"	<< endl;
	cout << "         Parallel version on GPU          "	<< endl;
	cout << "============================================"	<< endl;
    
    float* dev_h=new float[HSVsize];
    float* dev_s=new float[HSVsize];
    float* dev_v=new float[HSVsize];
    int * dev_rgb;
    int* newRGB = new int[rgb.size()];
    int* result = new int[rgb.size()];
    float* newV;


    HANDLE_ERROR(hipMalloc(&newRGB, rgb.size() * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_rgb, rgb.size() * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_h, HSVsize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&dev_s, HSVsize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&dev_v, HSVsize * sizeof(float)));
    HANDLE_ERROR(hipMalloc(&dev_histo, 256 * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_repartition, 256 * sizeof(int)));
    HANDLE_ERROR(hipMalloc(&newV, HSVsize * sizeof(float)));
    
    HANDLE_ERROR(hipMemcpy(dev_rgb, rgb.data(), rgb.size() * sizeof(int), hipMemcpyHostToDevice));
    nbBlocks=nbPixels/(1024)+1;
    int minGrid;
    int blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGrid,&blockSize,RGBtoHSV_GPU,0,nbPixels);
    int grid=(nbPixels+blockSize-1)/blockSize;
        cout<<minGrid<<" "<<blockSize<<" "<<grid<<" "<<nbBlocks<<endl;

    chrGPU.start();
    RGBtoHSV_GPU<<<nbBlocks,1024>>>(dev_rgb,dev_h,dev_s,dev_v,HSVsize); 
    chrGPU.stop();  
   
    cout << "-> RGB to HSV done : " << fixed << setprecision(2) << chrGPU.elapsedTime() << " ms" << endl << endl;
    totalTime += chrGPU.elapsedTime();    
    int * temp=new int[256];
    for(int i=0;i<256;i++){
        temp[i]=0;
    }
    HANDLE_ERROR(hipMemcpy(dev_histo, temp, 256*sizeof(int), hipMemcpyHostToDevice));
    nbBlocks=nbPixels/(1024*3)+1;
    chrGPU.start();
    computeHistogram_GPU_sharedMemoryVersion <<<32,32 >>>(dev_v, dev_histo,HSVsize);
    //computeHistogram_GPU <<<nbBlocks,1024 >>>(dev_v, dev_histo,HSVsize);
    chrGPU.stop();


    cout << "-> Compute histogram done : " << fixed << setprecision(2) << chrGPU.elapsedTime() << " ms" << endl << endl;
    totalTime += chrGPU.elapsedTime();


    chrGPU.start();
    repartition_GPU<<<1, 256 >>>(dev_histo,dev_repartition,256);
    chrGPU.stop();
    cout << "-> Compute Repartition done : " << fixed << setprecision(2) << chrGPU.elapsedTime() << " ms" << endl << endl;
    totalTime += chrGPU.elapsedTime();

   
    
       
    chrGPU.start();
    egalisation_GPU <<<nbBlocks, 256 >>>(dev_repartition,dev_v,newV,HSVsize);
    chrGPU.stop();


    cout << "-> Compute Egalisation done : " << fixed << setprecision(2) << chrGPU.elapsedTime() << " ms" << endl << endl;
    totalTime += chrGPU.elapsedTime();
    nbBlocks=nbPixels/(1024*3)+1;
    chrGPU.start();
    HSVtoRGB_GPU<<<nbBlocks,1024>>>(dev_h,dev_s,newV,newRGB,HSVsize);
    chrGPU.stop();
    float* test = new float[HSVsize];
    HANDLE_ERROR(hipMemcpy(test, newV, HSVsize*sizeof(float), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(result, newRGB, rgb.size() * sizeof(int), hipMemcpyDeviceToHost));
    cout << "-> HSV to FINAL RGB done : " << fixed << setprecision(2) << chrGPU.elapsedTime() << " ms" << endl << endl;
    totalTime += chrGPU.elapsedTime();
    cout << "-> ALL GPU DONE total time : " << fixed << setprecision(2) << totalTime<< " ms" << endl << endl;
    

    HANDLE_ERROR(hipFree(newV));
    HANDLE_ERROR(hipFree(dev_histo));
    HANDLE_ERROR(hipFree(dev_repartition));
    HANDLE_ERROR(hipFree(dev_rgb));
    HANDLE_ERROR(hipFree(dev_h));
    HANDLE_ERROR(hipFree(dev_s));
    HANDLE_ERROR(hipFree(dev_v));
    HANDLE_ERROR(hipFree(newRGB));
    
    return result;
}


__global__ void RGBtoHSV_GPU( int* dev_rgb,float* dev_h,float* dev_s,float* dev_v,int rgbSize){// 1threads =rgb
    int tid =blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < rgbSize) {
       float r,g,b;
       float cMax,cMin,delta;
        r=(float) dev_rgb[tid*3]/255;
        g=(float) dev_rgb[tid*3+1]/255;
        b=(float) dev_rgb[tid*3+2]/255;
        

        cMax=max(r,g);
        cMax=max(cMax,b);
        cMin=min(r,g);
        cMin=min(cMin,b);
        delta = cMax-cMin;
    
        // Calcul de H
        float hue=0;
        if(cMax == r){
            float left=((g-b)/delta);
            int quot=left/6;
            float mod=left-quot*6;
            hue=60 * mod;
        }else if (cMax == g){
            hue=60*(((b-r)/delta)+2);//Calcul a faire
        }
        else if(cMax== b){
           hue=60*(((r-g)/delta)+4);
        }
        else {
            hue=0.0;//Calcul a faire
        }
        if(hue<0){
            hue+=360;
        }
        dev_h[tid]=hue;
        // Calcul de S
        if (cMax>0.0){
            dev_s[tid]=delta/cMax;
        } 
        else {
            dev_s[tid]=0;
        }
        // Calcul de V
        dev_v[tid]=cMax;
        tid += gridDim.x * blockDim.x;
    
   }
}
__global__ void HSVtoRGB_GPU(float* h,float* s,float* newV,int* rgb,int HSVsize){
int tid =blockIdx.x * blockDim.x + threadIdx.x;
   while (tid < HSVsize) {
       
    float c,x,hue,m,rTemp,gTemp,bTemp;
    hue=h[tid];
    c= newV[tid]*s[tid];
    int quot=(int)((hue/60)/2);
    float fmod=hue/60-(quot*2.0);
    fmod-=1;
    if(fmod<0)
        fmod=-fmod;    
       x=c*(1-fmod);
       m= newV[tid]-c;
    if(hue>=0 && hue<60){
        rTemp=c;
        gTemp=x;
        bTemp=0;
    }
    else if(hue>=60 && hue<120){
        rTemp=x;
        gTemp=c;
        bTemp=0;
    }
    else if(hue>=120 && hue<180){
        rTemp=0;
        gTemp=c;
        bTemp=x;     
    }
    else if(hue>=180 && hue<240){
        rTemp=0;
        gTemp=x;
        bTemp=c;
    }
    else if(hue>=240 && hue<300){
        rTemp=x;
        gTemp=0;
        bTemp=c;
    }
    else{ //300 360
        rTemp=c;
        gTemp=0;
        bTemp=x;
    }
    rTemp=(rTemp+m)*255;
    gTemp=(gTemp+m)*255;
    bTemp=(bTemp+m)*255;
    
    rgb[tid*3]=int(rTemp);
    rgb[tid*3+1]=int(gTemp);
    rgb[tid*3+2]=int(bTemp);
        
    tid += gridDim.x * blockDim.x;
   }


}
__global__
void computeHistogram_GPU(const float* dev_v , int* dev_histo, const int v_size){
     int tid = blockIdx.x * blockDim.x + threadIdx.x;
   
     while (tid < v_size) {
         atomicAdd(&dev_histo[int(dev_v[tid] * 255)],1);
         tid += gridDim.x * blockDim.x;
     }
}

__global__
void computeHistogram_GPU_sharedMemoryVersion(const float* dev_v, int* dev_histo, const int v_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int histo[256];
    if (tid < 256) {
        histo[tid] = 0;
        dev_histo[tid] = 0;
    }
    __syncthreads();
    while (tid < v_size) {
        atomicAdd(&histo[int(dev_v[tid] * 255)], 1);
        tid += gridDim.x * blockDim.x;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int i = 0; i < 256; i++) {
            atomicAdd(&dev_histo[i], histo[i]);
        }
    }
}



__global__
void repartition_GPU(int* dev_histo,int* dev_repartition, int size) {
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int val;
    
    if (tid < size) {
        dev_repartition[tid] = dev_histo[tid];
        val = dev_histo[tid];
    }
    __syncthreads();
    for (int i = tid+1; i < size; i++) {
        atomicAdd(&dev_repartition[i],val);
    }
    
   
   
}

__global__
void egalisation_GPU(const int* dev_repartition, float* dev_v,  float* newV ,const int v_size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (tid < v_size) {
        newV[tid] = (255.0 / (256.0 * v_size)) * dev_repartition[int(dev_v[tid] * 255)];
        tid += gridDim.x * blockDim.x;
    }
    
}