#include "chronoGPU.hpp"
#include "common.hpp"
#include <iostream>

ChronoGPU::ChronoGPU() 
	: m_started( false ) {
	HANDLE_ERROR( hipEventCreate( &m_start ) );
	HANDLE_ERROR( hipEventCreate( &m_end ) );
}

ChronoGPU::~ChronoGPU() {
	if ( m_started ) {
		stop();
		std::cerr << "ChronoGPU::~ChronoGPU(): hrono wasn't turned off!" << std::endl; 
	}
	HANDLE_ERROR( hipEventDestroy( m_start ) );
	HANDLE_ERROR( hipEventDestroy( m_end ) );
}

void ChronoGPU::start() {
	if ( !m_started ) {
		HANDLE_ERROR( hipEventRecord( m_start, 0 ) );
		m_started = true;
	}
	else
		std::cerr << "ChronoGPU::start(): chrono is already started!" << std::endl;
}

void ChronoGPU::stop() {
	if ( m_started ) {
		HANDLE_ERROR( hipEventRecord( m_end, 0 ) );
		HANDLE_ERROR( hipEventSynchronize( m_end ) );
		m_started = false;
	}
	else
		std::cerr << "ChronoGPU::stop(): chrono wasn't started!" << std::endl;
}

float ChronoGPU::elapsedTime() { 
	float time = 0.f;
	HANDLE_ERROR( hipEventElapsedTime( &time, m_start, m_end ) );
	return time;
}
